#include "hip/hip_runtime.h"
// see doc: https://developer.nvidia.com/docs/drive/drive-os/6.0.9.1/public/drive-os-tensorrt/api-reference/docs/cpp/classnvinfer1_1_1_i_plugin_v2.html#a6a9cd7a410494f90b527a413adc84ce4

#include <iostream> // For std::cerr and std::endl

#include "ScatterNDPlugin.h"
#include <cassert>
#include <cstring> // for memcpy
#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"

// Define these in your .h or .cu file
#define SCATTERND_PLUGIN_NAME "ScatterND"
#define SCATTERND_PLUGIN_VERSION "1"
#define THREAD_NUM 1024  // Ensure this is set before using

namespace
{
template <typename T>
void write(char*& buffer, const T& val)
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

template <typename T>
void read(const char*& buffer, T& val)
{
    val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
}
} // anonymous namespace

template <typename T>
T readFromBuffer(const char*& buffer) {
    T val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
    return val;
}

namespace nvinfer1
{
namespace plugin
{
ScatterNDPlugin::ScatterNDPlugin(const std::string& name, const size_t outputShape[], const size_t inputShape[], DataType type)
: mLayerName(name), mDataType(type)
{
    mOutputSize[0] = outputShape[0];
    mOutputSize[1] = outputShape[1];
    mInputIndexSize[0] = inputShape[0];
    mInputIndexSize[1] = inputShape[1];
}

ScatterNDPlugin::ScatterNDPlugin(const std::string& name, const void* data, size_t length)
: mLayerName(name)
{
    const char* d = reinterpret_cast<const char*>(data);
    mDataType = readFromBuffer<DataType>(d);
    mOutputSize[0] = readFromBuffer<size_t>(d);
    mOutputSize[1] = readFromBuffer<size_t>(d);
    mInputIndexSize[0] = readFromBuffer<size_t>(d);
    mInputIndexSize[1] = readFromBuffer<size_t>(d);
}

int ScatterNDPlugin::getNbOutputs() const TRT_NOEXCEPT
{
    return 1;
}

Dims ScatterNDPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims) TRT_NOEXCEPT
{
    assert(index == 0);
    assert(nbInputDims == 2);
    return Dims3(inputs[0].d[0], inputs[0].d[1], 1); // Example modification
}

int ScatterNDPlugin::initialize() TRT_NOEXCEPT
{
    return 0;
}

void ScatterNDPlugin::terminate() TRT_NOEXCEPT
{
}

size_t ScatterNDPlugin::getWorkspaceSize(int) const TRT_NOEXCEPT
{
    return 0;
}

// DataType ScatterNDPlugin::getOutputDataType(int index, const DataType* inputTypes, int nbInputs) const noexcept override {
//     return inputTypes[index];  // Assuming the return type is based on the input type at the same index
// }

template <typename Dtype>
__global__ void _ScatterNDKernel(const Dtype *updata_input, const int *indicesInputPtr , Dtype* output,
        int channel_num, int max_index_num) {
    
    int idx_num = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx_num >= max_index_num) return;    
    
    int idx_output = indicesInputPtr[idx_num*2+1];
    if (idx_output < 0) return;
    
    for(int idx=0; idx < channel_num; idx++){
        output[idx_output*channel_num+idx] = updata_input[idx_num*channel_num+idx];
    }
}

int ScatterNDPlugin::enqueue(
        int32_t batchSize, 
        void const *const * inputs, 
        void *const * outputs, 
        void * workspace, 
        hipStream_t stream
    ) TRT_NOEXCEPT override
{
    int channel_num = mOutputSize[1];
    int max_index_num = mInputIndexSize[0];
    int totalElems = mOutputSize[0] * channel_num;

    dim3 blockSize(THREAD_NUM);
    dim3 gridsize((max_index_num + blockSize.x - 1) / blockSize.x);

    switch (mDataType) {
    case nvinfer1::DataType::kFLOAT:
        hipMemset(outputs[0], 0, totalElems * sizeof(float));
        _ScatterNDKernel<float><<<gridsize, blockSize, 0, stream>>>((float const*) inputs[2], (int32_t const*) inputs[1], (float*) outputs[0], channel_num, max_index_num);
        break;
    case nvinfer1::DataType::kHALF:
        hipMemset(outputs[0], 0, totalElems * sizeof(__half));
        _ScatterNDKernel<__half><<<gridsize, blockSize, 0, stream>>>((__half const*) inputs[2], (int32_t const*) inputs[1], (__half*) outputs[0], channel_num, max_index_num);
        break;
    default:
        std::cerr << "[ERROR]: Unsupported data type!" << std::endl;
        return -1;
    }
    return 0;
}

bool ScatterNDPlugin::supportsFormatCombination(int pos, const PluginTensorDesc* inOut, int nbInputs, int nbOutputs) const TRT_NOEXCEPT override {
    // Ensure the tensor format is kLINEAR, which is required.
    if (inOut[pos].format != TensorFormat::kLINEAR) {
        return false;
    }
    // Check if the data type is one of the supported formats.
    switch (inOut[pos].type) {
        case DataType::kFLOAT:
        case DataType::kINT32:
        case DataType::kHALF:
            return true;
        default:
            return false;
    }
}

void ScatterNDPlugin::serialize(void* buffer) const TRT_NOEXCEPT
{
    char* d = static_cast<char*>(buffer);
    write(d, mDataType);
    write(d, mOutputSize[0]);
    write(d, mOutputSize[1]);
    write(d, mInputIndexSize[0]);
    write(d, mInputIndexSize[1]);
}

size_t ScatterNDPlugin::getSerializationSize() const TRT_NOEXCEPT
{
    return sizeof(mDataType) + 4 * sizeof(size_t);
}

bool ScatterNDPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const TRT_NOEXCEPT
{
  return false;
}

bool ScatterNDPlugin::canBroadcastInputAcrossBatch(int inputIndex) const TRT_NOEXCEPT
{
  return false;
}

void ScatterNDPlugin::configurePlugin(const PluginTensorDesc* in, int32_t nbInput, const PluginTensorDesc* out, int32_t nbOutput) noexcept override {
    // Check the number of inputs and outputs first to avoid accessing out of bounds
    if (nbInput > 1 && nbOutput > 0) {
        // Configure internal buffer sizes based on the input and output tensor dimensions
        mOutputSize[0] = out[0].dims.d[0];
        mOutputSize[1] = out[0].dims.d[1];
        mInputIndexSize[0] = in[1].dims.d[0];
        mInputIndexSize[1] = in[1].dims.d[1];
    }
}

const char* ScatterNDPlugin::getPluginType() const TRT_NOEXCEPT
{
    return SCATTERND_PLUGIN_NAME;
}

const char* ScatterNDPlugin::getPluginVersion() const TRT_NOEXCEPT
{
    return SCATTERND_PLUGIN_VERSION;
}

void ScatterNDPlugin::destroy() TRT_NOEXCEPT
{
    delete this;
}

IPluginV2Ext* ScatterNDPlugin::clone() const TRT_NOEXCEPT override {
    ScatterNDPlugin* clonedPlugin = new ScatterNDPlugin(mLayerName, mOutputSize, mInputIndexSize, mDataType);
    clonedPlugin->setPluginNamespace(mNamespace.c_str());
    return clonedPlugin;  // Cast is not needed if ScatterNDPlugin is derived from IPluginV2Ext
}

void ScatterNDPlugin::setPluginNamespace(const char* libNamespace) TRT_NOEXCEPT
{
    mNamespace = libNamespace;
}

const char* ScatterNDPlugin::getPluginNamespace() const TRT_NOEXCEPT
{
    return mNamespace.c_str();
}

ScatterNDSamplePluginCreator::ScatterNDSamplePluginCreator()
{
    mPluginAttributes.emplace_back(PluginField("output_shape", nullptr, PluginFieldType::kINT32, 2));
    mPluginAttributes.emplace_back(PluginField("index_shape", nullptr, PluginFieldType::kINT32, 2));
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields = mPluginAttributes.data();
}

const char* ScatterNDSamplePluginCreator::getPluginName() const TRT_NOEXCEPT
{
    return SCATTERND_PLUGIN_NAME;
}

const char* ScatterNDSamplePluginCreator::getPluginVersion() const TRT_NOEXCEPT
{
    return SCATTERND_PLUGIN_VERSION;
}

const PluginFieldCollection* ScatterNDSamplePluginCreator::getFieldNames() TRT_NOEXCEPT
{
    return &mFC;
}

IPluginV2Ext* ScatterNDSamplePluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) TRT_NOEXCEPT
{
    const PluginField* fields = fc->fields;
    size_t outputShapeArray[2] = {0, 0};
    size_t indexShapeArray[2] = {0, 0};
    DataType dataType = DataType::kFLOAT;  // Default data type

    for (int i = 0; i < fc->nbFields; i++)
    {
        std::string fieldName(fields[i].name);
        if (fieldName == "output_shape")
        {
            const int32_t* shape = static_cast<const int32_t*>(fields[i].data);
            outputShapeArray[0] = shape[0];
            outputShapeArray[1] = shape[1];
        }
        else if (fieldName == "index_shape")
        {
            const int32_t* shape = static_cast<const int32_t*>(fields[i].data);
            indexShapeArray[0] = shape[0];
            indexShapeArray[1] = shape[1];
        }
    }

    ScatterNDPlugin* plugin = new ScatterNDPlugin(name, outputShapeArray, indexShapeArray, dataType);
    plugin->setPluginNamespace(mNamespace.c_str());
    return plugin;
}

void ScatterNDSamplePluginCreator::setPluginNamespace(const char* libNamespace) TRT_NOEXCEPT
{
    mNamespace = libNamespace;
}

const char* ScatterNDSamplePluginCreator::getPluginNamespace() const TRT_NOEXCEPT
{
    return mNamespace.c_str();
}

REGISTER_TENSORRT_PLUGIN(ScatterNDSamplePluginCreator);

} // namespace plugin
} // namespace nvinfer1
